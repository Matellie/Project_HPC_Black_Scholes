
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <numeric>

#include <hiprand/hiprand_kernel.h>


__global__ void setup_kernel(hiprandStatePhilox4_32_10_t* state, long int random_thing) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence number, no offset */
    //curand_init(1234, id, 0, &state[id]);
    hiprand_init((1234 + random_thing * threadIdx.x * blockIdx.x * blockDim.x) % 14569, id, 0, &state[id]);
}


__global__ void generate_monte_carlo_bs(
    hiprandStatePhilox4_32_10_t* state,
    long int nbSim,
    int lengthSim,
    double* result,
    double K,
    double S0,
    double T,
    double r,
    double sigma
) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandStatePhilox4_32_10_t localState = state[id];

    double dt = T / lengthSim;
    double drift = (r - 0.5 * sigma * sigma) * dt;
    double diffusion = sigma * sqrt(dt);

    for (int i=id; i<nbSim; i+=blockDim.x*gridDim.x) {
        double ST = S0;
        for (int j=0; j<lengthSim; ++j) {
            double r = hiprand_normal_double(&localState);
            ST *= exp(drift + diffusion * r);
            state[id] = localState;
        }
        if (ST > K) {
            result[i] = ST - K;
        } else {
            result[i] = 0;
        }
    }
}


double monteCarloBlackScholes(
    double S0,
    double K,
    double T,
    double r,
    double sigma,
    long int nbSim,
    int lengthSim
) {
    const unsigned int threadsPerBlock = 256;
    const unsigned int blockCount = 100;
    hiprandStatePhilox4_32_10_t* devPHILOXStates;
    hipMalloc((void**)&devPHILOXStates, threadsPerBlock * blockCount * sizeof(hiprandStatePhilox4_32_10_t));

    double* result_gpu;
    double* result_cpu = new double[nbSim];
    hipMalloc(&result_gpu, nbSim * sizeof(double));

    auto now = std::chrono::high_resolution_clock::now();
    long int nanos = (long int)std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
    setup_kernel<<<blockCount, threadsPerBlock>>>(devPHILOXStates, nanos);

    generate_monte_carlo_bs<<<blockCount, threadsPerBlock>>>(
        devPHILOXStates, nbSim, lengthSim, result_gpu, K, S0, T, r, sigma
    );

    hipMemcpy(result_cpu, result_gpu, nbSim * sizeof(double), hipMemcpyDeviceToHost);

    double sum = std::accumulate(result_cpu, result_cpu + nbSim, 0.0);
    double mean = sum / nbSim;

    return mean;
}


int main(int argc, char** argv) {
    // Parameters
    double S0 = 100.0;      // Initial stock price
    double K = 150.0;       // Strike price
    double T = 1.0;         // Time to maturity (1 year)
    double r = 0.05;        // Risk-free rate (5%)
    double sigma = 0.2;     // Volatility (20%)
    long int nbSim = 1e6;   // Number of simulation paths
    int lengthSim = 100;    // Number of time intervals

    // Read command line arguments.
    for (int i = 0; i < argc; i++) {
        if (strcmp(argv[i], "--nbSim") == 0) {
            nbSim = atol(argv[++i]);
        }
        else if (strcmp(argv[i], "--lengthSim") == 0) {
            lengthSim = atoi(argv[++i]);
        }
        else if (strcmp(argv[i], "--S0") == 0) {
            S0 = strtod(argv[++i], nullptr);
        }
        else if (strcmp(argv[i], "--K") == 0) {
            K = strtod(argv[++i], nullptr);
        }
        else if (strcmp(argv[i], "--T") == 0) {
            T = strtod(argv[++i], nullptr);
        }
        else if (strcmp(argv[i], "--r") == 0) {
            r = strtod(argv[++i], nullptr);
        }
        else if (strcmp(argv[i], "--sigma") == 0) {
            sigma = strtod(argv[++i], nullptr);
        }
        else if (strcmp(argv[i], "--h") == 0 || strcmp(argv[i], "--help") == 0) {
            std::cout
                << "Options:\n"
                << "  --S0 <double>: Initial stock price (default 100.0)\n"
                << "  --K <double>: Strike price (default 100.0)\n"
                << "  --T <double>: Time to maturity (default 1.0)\n"
                << "  --r <double>: Risk-free rate (default 0.05)\n"
                << "  --sigma <double>: Volatility (default 0.2)\n"
                << "  --nbSim <int>: Number of simulation paths (default 1e6)\n"
                << "  --lengthSim <int>: Number of time intervals (default 10)\n"
                << "  --help (-h): Print this message\n";
            return 0;
        }
    }

    auto start = std::chrono::high_resolution_clock::now();
    double price = monteCarloBlackScholes(S0, K, T, r, sigma, nbSim, lengthSim);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Option Price: " << price << std::endl;
    std::cout << "Elapsed Time: " << elapsed.count() << " seconds" << std::endl;

    return 0;
}
